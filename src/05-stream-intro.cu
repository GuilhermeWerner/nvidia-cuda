
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

__global__ void printNumber(int number)
{
    printf("%d\n", number);
}

int main()
{
    for (int i = 0; i < 5; ++i)
    {
        hipStream_t stream;                 // CUDA streams are of type `cudaStream_t`.
        hipStreamCreate(&stream);           // Note that a pointer must be passed to `cudaCreateStream`.
        printNumber<<<1, 2, 0, stream>>>(i); // `stream` is passed as 4th EC argument.
        hipStreamDestroy(stream);
    }
    hipDeviceSynchronize();
}
